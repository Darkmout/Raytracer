#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Model.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <gl/glut.h>

//  Avoid showing up the console window
#pragma comment(linker,"/subsystem:\"windows\" /entry:\"mainCRTStartup\"")

//  constants representing the window size
#define WINDOW_WIDTH 512
#define WINDOW_HEIGHT 512

//  Initialization
void init ();

//  Callback functions
void display (void);
void mouse (int button, int state, int x, int y);
void keyboard (unsigned char key, int x, int y);

//  Support Functions
void centerOnScreen ();

//  define the window position on screen
int window_x;
int window_y;

//  variable representing the window title
char *window_title = "Image Generator";

//  Tells whether to display the window full screen or not
//  Press Alt + Esc to exit a full screen.
int full_screen = 0;

//  Generates a random image...
void generateImage ();

//  Represents the pixel buffer in memory
GLubyte buffer[WINDOW_WIDTH][WINDOW_HEIGHT][3];




//-------------------------------------------------------------------------
//  Set OpenGL program initial state.
//-------------------------------------------------------------------------
void init ()
{	
	glClearColor (0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_FLAT);
	glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
	srand (time(NULL));
	generateImage();
}

//-------------------------------------------------------------------------
//  This function is passed to glutDisplayFunc in order to display 
//	OpenGL contents on the window.
//-------------------------------------------------------------------------
void display (void)
{
	glClear(GL_COLOR_BUFFER_BIT);
	glDrawPixels(WINDOW_WIDTH, WINDOW_HEIGHT, GL_RGB,
		GL_UNSIGNED_BYTE, buffer);
	glutSwapBuffers ();
}



//-------------------------------------------------------------------------
//  This function is passed to the glutMouseFunc and is called 
//  whenever the mouse is clicked.
//-------------------------------------------------------------------------
void mouse (int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		generateImage ();
		glutPostRedisplay ();
	}
}

//-------------------------------------------------------------------------
//  This function is passed to the glutKeyboardFunc and is called 
//  whenever the user hits a key.
//-------------------------------------------------------------------------
void keyboard (unsigned char key, int x, int y)
{
	switch (key)
	{
	case 'g':
		generateImage ();
		glutPostRedisplay ();
		break;
	case 27:
		exit (0);
	}
}

//-------------------------------------------------------------------------
//  This function sets the window x and y coordinates
//  such that the window becomes centered
//-------------------------------------------------------------------------
void centerOnScreen ()
{
	window_x = (glutGet (GLUT_SCREEN_WIDTH) - WINDOW_WIDTH)/2;
	window_y = (glutGet (GLUT_SCREEN_HEIGHT) - WINDOW_HEIGHT)/2;
}

void CheckCudaError(hipError_t cudaStatus)
{
	if(cudaStatus != hipSuccess)
	{
		printf(hipGetErrorString(cudaStatus));
		exit(1);
	}
}

__device__ Point CrossProduct(Point v1, Point v2)
{
	Point result;
	result.x = v1.y * v2.z - v1.z * v2.y;
	result.y = v1.z * v2.x - v1.x * v2.z;
	result.z = v1.x * v2.y - v1.y * v1.x;
	return result;
}
__device__ Vector getVecteur (Point origin, Point B)
{
	Vector vector = {
		origin,
		{
			B.x - origin.x,
				B.y - origin.y,
				B.z - origin.z
		}
	};
	return vector;
}

__device__ Vector GetNormalVecteur(Face face)
{
	Vector result;
	result.origin = face.A;
	result.direction = CrossProduct(getVecteur(face.A, face.B).direction, getVecteur(face.A, face.C).direction);
	return result;
}


__device__ bool Intersect(Vector vector, Face face)
{
	//I0 + (I1 - I0)t = P+ (P1 - P0)u + (P2 - P0)v;
	face.normal = GetNormalVecteur(face);
	float o1 = vector.origin.x;
	float o2 = vector.origin.y;
	float o3 = vector.origin.z;

	float d1 = vector.direction.x;
	float d2 = vector.direction.y;
	float d3 = vector.direction.z;

	float x = face.normal.direction.x;
	float y = face.normal.direction.y;
	float z = face.normal.direction.z;


	float a = face.A.x;
	float b = face.A.y;
	float c = face.A.z;
	float d = - (face.a * face.normal.direction.x) - (face.b * face.normal.direction.y) - (face.c * face.normal.direction.z);

	float t = (x * o1 - x * a + y * o2 - y * b + z * o3 - z * c) / (- x * d1 - y * d2 - z * d3);
	printf("%f \n", t);
	Point result;
	result.x = o1 + d1 * t;
	result.y = o2 + d2 * t;
	result.z = o3 + d3 * t;

	return (result.y > face.A.y && result.z > face.A.z);
}


__device__ Point FaceToWorld(Point local, Face face)
{

	Point global = {
		face.A.x*local.x*local.y + face.B.x*(1-local.x)*(local.y) + face.C.x*(local.x)*(1-local.y) + face.D.x*(1-local.x)*(1-local.y),
		face.A.y*local.x*local.y + face.B.y*(1-local.x)*(local.y) + face.C.y*(local.x)*(1-local.y) + face.D.y*(1-local.x)*(1-local.y),
		face.A.z*local.x*local.y + face.B.z*(1-local.x)*(local.y) + face.C.z*(local.x)*(1-local.y) + face.D.z*(1-local.x)*(1-local.y)
	};

	return global;

}



//kernel
__global__ void RayKernel(uchar4* const outputImageRGBA,Camera camera , Face face, int numRows, int numCols)
{
	//computing the thread index
	const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);
	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;

	//computing the coordinate of the pixel from the screen to the world
	Point pixel = { 1.0 - (float)thread_2D_pos.x / (float)numCols,  1.0 - (float)thread_2D_pos.y / (float)numRows, 0};
	Point pixelCoordinate = FaceToWorld(pixel, camera.plan);

	//coputing the Vector
	Vector vector = getVecteur(camera.origin, pixelCoordinate);


	//computing the intersection
	if(Intersect(vector, face))
		outputImageRGBA[thread_1D_pos] = make_uchar4(255,255,255, 255);
	else
		outputImageRGBA[thread_1D_pos] = make_uchar4(0,0,0, 255);

}

//-------------------------------------------------------------------------
//  Generate new image with random colors
//-------------------------------------------------------------------------
void generateImage ()
{


	const dim3 blockSize(8 , 8);
	const dim3 gridSize (WINDOW_WIDTH / blockSize.x + 1, WINDOW_HEIGHT / blockSize.y + 1);

	//image
	uchar4 *d_outputImageRGBA, *h_outputImageRGBA;
	h_outputImageRGBA = (uchar4*)malloc(  sizeof(uchar4) * WINDOW_WIDTH * WINDOW_HEIGHT);
	CheckCudaError(hipMalloc(&d_outputImageRGBA,   sizeof(uchar4) * WINDOW_WIDTH * WINDOW_HEIGHT));

	//elements

	Camera camera = { 
		{0, 0, 0},

		{	
			1, 0.5, 0.5,
				1, 0.5, -0.5,
				1, -0.5, -0.5,
				1, -0.5, 0.5,
		}

	};

	Face face = {	
		2, 0.5, 0.5,
		2, 0.5, -0.5,
		2, -0.5, -0.5,
		2, -0.5, 0.5,
	};


	RayKernel<<<gridSize, blockSize>>>(d_outputImageRGBA, camera, face, WINDOW_HEIGHT, WINDOW_WIDTH);	hipDeviceSynchronize(); CheckCudaError(hipGetLastError());

	CheckCudaError(hipMemcpy(h_outputImageRGBA, d_outputImageRGBA, sizeof(uchar4) * WINDOW_WIDTH * WINDOW_HEIGHT, hipMemcpyDeviceToHost));

	int i, j;
	for (i = 0; i < WINDOW_WIDTH; i++) 
	{
		for (j = 0; j < WINDOW_HEIGHT; j++)
		{
			buffer[i][j][0] = (GLubyte) (h_outputImageRGBA[i + j* WINDOW_HEIGHT].x);
			buffer[i][j][1] = (GLubyte) (h_outputImageRGBA[i + j* WINDOW_HEIGHT].y);
			buffer[i][j][2] = (GLubyte) (h_outputImageRGBA[i + j* WINDOW_HEIGHT].z);
		}
	}
}


//-------------------------------------------------------------------------
//  Program Main method.
//-------------------------------------------------------------------------
void main (int argc, char **argv)
{
	//  Connect to the windowing system
	glutInit(&argc, argv);

	//  create a window with the specified dimensions
	glutInitWindowSize (WINDOW_WIDTH, WINDOW_HEIGHT);

	//  Set the window x and y coordinates such that the 
	//  window becomes centered
	centerOnScreen ();

	//  Position Window
	glutInitWindowPosition (window_x, window_y);

	//  Set Display mode
	glutInitDisplayMode (GLUT_RGB | GLUT_DOUBLE);

	//  Create window with the specified title
	glutCreateWindow (window_title);

	//  View in full screen if the full_screen flag is on
	if (full_screen)
		glutFullScreen ();

	//  Set OpenGL program initial state.
	init();

	// Set the callback functions
	glutDisplayFunc (display);
	glutKeyboardFunc (keyboard);
	glutMouseFunc (mouse);

	//  Start GLUT event processing loop
	glutMainLoop();
}
