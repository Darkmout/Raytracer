#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Camera.h"
#include "Point.h"
#include "Plane.h"
#include "Vec3.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <gl/glut.h>

//  Avoid showing up the console window
#pragma comment(linker,"/subsystem:\"windows\" /entry:\"mainCRTStartup\"")

//  constants representing the window size
#define WINDOW_WIDTH 512
#define WINDOW_HEIGHT 512

//  Initialization
void init ();

//  Callback functions
void display (void);
void mouse (int button, int state, int x, int y);
void keyboard (unsigned char key, int x, int y);

//  Support Functions
void centerOnScreen ();

//  define the window position on screen
int window_x;
int window_y;

//  variable representing the window title
char *window_title = "Image Generator";

//  Tells whether to display the window full screen or not
//  Press Alt + Esc to exit a full screen.
int full_screen = 0;

//  Generates a random image...
void generateImage ();

//  Represents the pixel buffer in memory
GLubyte buffer[WINDOW_WIDTH][WINDOW_HEIGHT][3];




//-------------------------------------------------------------------------
//  Set OpenGL program initial state.
//-------------------------------------------------------------------------
void init ()
{	
	glClearColor (0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_FLAT);
	glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
	srand (time(NULL));
	generateImage();
}

//-------------------------------------------------------------------------
//  This function is passed to glutDisplayFunc in order to display 
//	OpenGL contents on the window.
//-------------------------------------------------------------------------
void display (void)
{
	glClear(GL_COLOR_BUFFER_BIT);
	glDrawPixels(WINDOW_WIDTH, WINDOW_HEIGHT, GL_RGB,
		GL_UNSIGNED_BYTE, buffer);
	glutSwapBuffers ();
}



//-------------------------------------------------------------------------
//  This function is passed to the glutMouseFunc and is called 
//  whenever the mouse is clicked.
//-------------------------------------------------------------------------
void mouse (int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		generateImage ();
		glutPostRedisplay ();
	}
}

//-------------------------------------------------------------------------
//  This function is passed to the glutKeyboardFunc and is called 
//  whenever the user hits a key.
//-------------------------------------------------------------------------
void keyboard (unsigned char key, int x, int y)
{
	switch (key)
	{
	case 'g':
		generateImage ();
		glutPostRedisplay ();
		break;
	case 27:
		exit (0);
	}
}

//-------------------------------------------------------------------------
//  This function sets the window x and y coordinates
//  such that the window becomes centered
//-------------------------------------------------------------------------
void centerOnScreen ()
{
	window_x = (glutGet (GLUT_SCREEN_WIDTH) - WINDOW_WIDTH)/2;
	window_y = (glutGet (GLUT_SCREEN_HEIGHT) - WINDOW_HEIGHT)/2;
}

void CheckCudaError(hipError_t cudaStatus)
{
	if(cudaStatus != hipSuccess)
	{
		printf(hipGetErrorString(cudaStatus));
		exit(1);
	}
}



//kernel
__global__ void RayKernel(uchar4* const outputImageRGBA,Camera camera , Plane plane, int numRows, int numCols)
{
	//computing the thread index
	const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;


	Ray ray = camera.GetRay(thread_2D_pos.x, thread_2D_pos.y);

	//computing the intersection
	if(plane.Intersect(ray))
		outputImageRGBA[thread_1D_pos] = make_uchar4(255,255,255, 255);
	else
		outputImageRGBA[thread_1D_pos] = make_uchar4(100,0,0, 255);

}

//-------------------------------------------------------------------------
//  Generate new image with random colors
//-------------------------------------------------------------------------
void generateImage ()
{


	const dim3 blockSize(16 , 16);
	const dim3 gridSize (WINDOW_WIDTH / blockSize.x + 1, WINDOW_HEIGHT / blockSize.y + 1);

	//image
	uchar4 *d_outputImageRGBA, *h_outputImageRGBA;
	h_outputImageRGBA = (uchar4*)malloc(sizeof(uchar4) * WINDOW_WIDTH * WINDOW_HEIGHT);
	CheckCudaError(hipMalloc(&d_outputImageRGBA,   sizeof(uchar4) * WINDOW_WIDTH * WINDOW_HEIGHT));

	//elements

	Camera camera = Camera(WINDOW_WIDTH, WINDOW_HEIGHT);

	Plane plane = Plane(
		Vec3(3,0.5,0.5),
		Vec3(3,0.5,-0.5),
		Vec3(3,-0.5,-0.5),
		Vec3(3,-0.5,0.5)
		);

	RayKernel<<<gridSize, blockSize>>>(d_outputImageRGBA, camera, plane, WINDOW_HEIGHT, WINDOW_WIDTH);	hipDeviceSynchronize(); CheckCudaError(hipGetLastError());

	CheckCudaError(hipMemcpy(h_outputImageRGBA, d_outputImageRGBA, sizeof(uchar4) * WINDOW_WIDTH * WINDOW_HEIGHT, hipMemcpyDeviceToHost));

	int i, j;
	for (i = 0; i < WINDOW_WIDTH; i++) 
	{
		for (j = 0; j < WINDOW_HEIGHT; j++)
		{
			buffer[i][j][0] = (GLubyte) (h_outputImageRGBA[i + j* WINDOW_HEIGHT].x);
			buffer[i][j][1] = (GLubyte) (h_outputImageRGBA[i + j* WINDOW_HEIGHT].y);
			buffer[i][j][2] = (GLubyte) (h_outputImageRGBA[i + j* WINDOW_HEIGHT].z);
		}
	}
}


//-------------------------------------------------------------------------
//  Program Main method.
//-------------------------------------------------------------------------
int main (int argc, char **argv)
{
	//  Connect to the windowing system
	glutInit(&argc, argv);

	//  create a window with the specified dimensions
	glutInitWindowSize (WINDOW_WIDTH, WINDOW_HEIGHT);

	//  Set the window x and y coordinates such that the 
	//  window becomes centered
	centerOnScreen ();

	//  Position Window
	glutInitWindowPosition (window_x, window_y);

	//  Set Display mode
	glutInitDisplayMode (GLUT_RGB | GLUT_DOUBLE);

	//  Create window with the specified title
	glutCreateWindow (window_title);

	//  View in full screen if the full_screen flag is on
	if (full_screen)
		glutFullScreen ();

	//  Set OpenGL program initial state.
	init();

	// Set the callback functions
	glutDisplayFunc (display);
	glutKeyboardFunc (keyboard);
	glutMouseFunc (mouse);

	//  Start GLUT event processing loop
	glutMainLoop();

	return 0;
}