#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
//#include "cuda_gl_interop.h"
//#include "hip/hip_runtime.h"

#include "Camera.h"
#include "Plane.h"
#include "Vec3.h"
#include "Model.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string>
#include <gl/glut.h>

//  Avoid showing up the console window
//#pragma comment(linker,"/subsystem:\"windows\" /entry:\"mainCRTStartup\"")

//  constants representing the window size
#define WINDOW_WIDTH 512
#define WINDOW_HEIGHT 512

//the scene
Model scene;

//  Initialization
void init ();

//  Callback functions
void display (void);
void mouse (int button, int state, int x, int y);
void keyboard (unsigned char key, int x, int y);

//  Support Functions
void centerOnScreen ();

//  define the window position on screen
int window_x;
int window_y;

//  variable representing the window title
char *window_title = "Image Generator";

//  Tells whether to display the window full screen or not
//  Press Alt + Esc to exit a full screen.
int full_screen = 0;

//  Generates a random image...
void generateImage ();

//  Represents the pixel buffer in memory
GLubyte buffer[WINDOW_WIDTH][WINDOW_HEIGHT][3];




//-------------------------------------------------------------------------
//  Set OpenGL program initial state.
//-------------------------------------------------------------------------
void init ()
{	
	glClearColor (0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_FLAT);
	glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
	srand (time(NULL));
	generateImage();
}

//-------------------------------------------------------------------------
//  This function is passed to glutDisplayFunc in order to display 
//	OpenGL contents on the window.
//-------------------------------------------------------------------------
void display (void)
{
	glClear(GL_COLOR_BUFFER_BIT);
	glDrawPixels(WINDOW_WIDTH, WINDOW_HEIGHT, GL_RGB,
		GL_UNSIGNED_BYTE, buffer);
	glutSwapBuffers ();
}



//-------------------------------------------------------------------------
//  This function is passed to the glutMouseFunc and is called 
//  whenever the mouse is clicked.
//-------------------------------------------------------------------------
void mouse (int button, int state, int x, int y)
{
	if (state == GLUT_DOWN)
	{
		generateImage ();
		glutPostRedisplay ();
	}
}

//-------------------------------------------------------------------------
//  This function is passed to the glutKeyboardFunc and is called 
//  whenever the user hits a key.
//-------------------------------------------------------------------------
void keyboard (unsigned char key, int x, int y)
{
	switch (key)
	{
	case 'g':
		generateImage ();
		glutPostRedisplay ();
		break;
	case 27:
		exit (0);
	}
}

//-------------------------------------------------------------------------
//  This function sets the window x and y coordinates
//  such that the window becomes centered
//-------------------------------------------------------------------------
void centerOnScreen ()
{
	window_x = (glutGet (GLUT_SCREEN_WIDTH) - WINDOW_WIDTH)/2;
	window_y = (glutGet (GLUT_SCREEN_HEIGHT) - WINDOW_HEIGHT)/2;
}

void CheckCudaError(hipError_t cudaStatus)
{
	if(cudaStatus != hipSuccess)
	{
		printf(hipGetErrorString(cudaStatus));
		exit(1);
	}
}



//kernel
__global__ void RayKernel(uchar4* const outputImageRGBA,Camera camera , Plane* scene, int sceneCount, int numRows, int numCols)
{
	//computing the thread index
	const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;


	Ray ray = camera.GetRay(thread_2D_pos.x, thread_2D_pos.y);
	//printf("thread [%d,%d], rayDirectio %f,%f,%f", thread_2D_pos.x, thread_2D_pos.y, ray.Direction.x, ray.Direction.y,ray.Direction.z);
	//computing the intersection
	bool intersect = false;
	for(int i = 0; i < sceneCount; i++)
	{
		if(scene[i].Intersect(ray))
		intersect = true;
	}

	if(intersect)
		outputImageRGBA[thread_1D_pos] = make_uchar4(255,255,255, 255);
	else
		outputImageRGBA[thread_1D_pos] = make_uchar4(0,0,0, 255);

	//outputImageRGBA[thread_1D_pos] = make_uchar4(((ray.Direction.x + 1) / 2) * 255,((ray.Direction.x + 1) / 2)  * 255, ((ray.Direction.x + 1) / 2)  * 255, 255);
}

//-------------------------------------------------------------------------
//  Generate new image with random colors
//-------------------------------------------------------------------------
void generateImage ()
{


	const dim3 blockSize(16 , 16);
	const dim3 gridSize (WINDOW_WIDTH / blockSize.x + 1, WINDOW_HEIGHT / blockSize.y + 1);

	//initialisation
	uchar4 *d_outputImageRGBA, *h_outputImageRGBA;
	Plane *d_scene;
	h_outputImageRGBA = (uchar4*)malloc(sizeof(uchar4) * WINDOW_WIDTH * WINDOW_HEIGHT);
	CheckCudaError(hipMalloc(&d_outputImageRGBA,   sizeof(uchar4) * WINDOW_WIDTH * WINDOW_HEIGHT));

	CheckCudaError(hipMalloc(&d_scene,   sizeof(Plane) * scene.Planes.size()));
	CheckCudaError(hipMemcpy(d_scene, &scene.Planes[0], sizeof(Plane) * scene.Planes.size(), hipMemcpyHostToDevice));

	Camera camera = Camera(WINDOW_WIDTH, WINDOW_HEIGHT);

	RayKernel<<<gridSize, blockSize>>>(d_outputImageRGBA, camera, d_scene, scene.Planes.size(), WINDOW_HEIGHT, WINDOW_WIDTH);	hipDeviceSynchronize(); CheckCudaError(hipGetLastError());

	CheckCudaError(hipMemcpy(h_outputImageRGBA, d_outputImageRGBA, sizeof(uchar4) * WINDOW_WIDTH * WINDOW_HEIGHT, hipMemcpyDeviceToHost));

	int i, j;
	for (i = 0; i < WINDOW_WIDTH; i++) 
	{
		for (j = 0; j < WINDOW_HEIGHT; j++)
		{
			buffer[i][j][0] = (GLubyte) (h_outputImageRGBA[i + j* WINDOW_HEIGHT].x);
			buffer[i][j][1] = (GLubyte) (h_outputImageRGBA[i + j* WINDOW_HEIGHT].y);
			buffer[i][j][2] = (GLubyte) (h_outputImageRGBA[i + j* WINDOW_HEIGHT].z);
		}
	}
}


//-------------------------------------------------------------------------
//  Program Main method.
//-------------------------------------------------------------------------
int main (int argc, char* argv[])
{
	scene = Model(std::string(argv[argc-1]));

	//TODO : use Cuda Interop
	//  Connect to the windowing system
	glutInit(&argc, argv);

	//  create a window with the specified dimensions
	glutInitWindowSize (WINDOW_WIDTH, WINDOW_HEIGHT);

	//  Set the window x and y coordinates such that the 
	//  window becomes centered
	centerOnScreen ();

	//  Position Window
	glutInitWindowPosition (window_x, window_y);

	//  Set Display mode
	glutInitDisplayMode (GLUT_RGB | GLUT_DOUBLE);

	//  Create window with the specified title
	glutCreateWindow (window_title);

	//  View in full screen if the full_screen flag is on
	if (full_screen)
		glutFullScreen ();

	//  Set OpenGL program initial state.
	init();

	// Set the callback functions
	glutDisplayFunc (display);
	glutKeyboardFunc (keyboard);
	glutMouseFunc (mouse);

	//  Start GLUT event processing loop
	glutMainLoop();

	return 0;
}